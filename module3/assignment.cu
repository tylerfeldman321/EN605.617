//Based on the work of Andrew Krepps

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <random>
#include <iostream>
#include <chrono>
#include <algorithm>
using namespace std;

#define ARRAY_SIZE 20480
#define ARRAY_SIZE_IN_BYTES (sizeof(int) * (ARRAY_SIZE))

int cpu_a[ARRAY_SIZE];
int cpu_b[ARRAY_SIZE];
int cpu_result[ARRAY_SIZE];
bool verbose = false;

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

__global__
void add(int *result, int *a, int *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
	// printf("Result[%d]: %d = %d + %d\n", i, result[i], a[i], b[i]);
  }
}

__global__
void subtract(int *result, int *a, int *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] - b[i];
	// printf("Result[%d]: %d = %d + %d\n", i, result[i], a[i], b[i]);
  }
}

__global__
void multiply(int *result, int *a, int *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] * b[i];
	// printf("Result[%d]: %d = %d + %d\n", i, result[i], a[i], b[i]);
  }
}

__global__
void mod(int *result, int *a, int *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] % b[i];
	// printf("Result[%d]: %d = %d mod %d\n", i, result[i], a[i], b[i]);
  }
}

__global__
void branchingKernel(int *result, int *a, int *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  for(int i = index; i < N; i += stride)
  {
	if (i % 2 == 0) {
    	result[i] = a[i] + b[i];
	} else {
		result[i] = a[i] * b[i];
	}
  }
}


void initCpuArrays() {
	// Initializes cpu_a data to 0...N and cpu_b data to random numbers from 0-3 inclusive
	for (int i = 0; i < ARRAY_SIZE; i++) {
		cpu_a[i] = i;
	}
    random_device rd;
    mt19937 gen(rd());
    uniform_int_distribution<> distrib(0, 3);
	for (int i = 0; i < ARRAY_SIZE; i++) {
		cpu_b[i] = distrib(gen);
	}
}

void performMathOperations(int numBlocks, int blockSize, int totalThreads, std::string operation) {
	// performMathOperations()
	//  takes # blocks for kernel, block size (threads/block), total threads, and the math operation to do (add, subtract, multiply, or mod) and performs the operation
	if (verbose) {
		printf("----- Math Operations -----\n");
		printf("Op: %s, Array length: %d, Array bytes: %d, "
			"Blocks: %d, Threads/block: %d, Total threads: %d\n",
			operation.c_str(), (int)ARRAY_SIZE, (int)ARRAY_SIZE_IN_BYTES, 
			numBlocks, blockSize, totalThreads);
	}
	initCpuArrays();

	int *gpu_a;
	int *gpu_b;
	int *gpu_result;
	checkCuda( hipMalloc((void **)&gpu_a, ARRAY_SIZE_IN_BYTES) );
	checkCuda( hipMalloc((void **)&gpu_b, ARRAY_SIZE_IN_BYTES) );
	checkCuda( hipMalloc((void **)&gpu_result, ARRAY_SIZE_IN_BYTES) );

	hipMemcpy( gpu_a, cpu_a, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice );
	hipMemcpy( gpu_b, cpu_b, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice );

	// Perform and time the operation, synchronizing before stopping the timer
	auto start = std::chrono::high_resolution_clock::now();
	if (operation == "add") {
		add<<<numBlocks, blockSize>>>(gpu_result, gpu_a, gpu_b, ARRAY_SIZE);
	} else if (operation == "subtract") {
		subtract<<<numBlocks, blockSize>>>(gpu_result, gpu_a, gpu_b, ARRAY_SIZE);
	} else if (operation == "multiply") {
		multiply<<<numBlocks, blockSize>>>(gpu_result, gpu_a, gpu_b, ARRAY_SIZE);
	} else if (operation == "mod") {
		mod<<<numBlocks, blockSize>>>(gpu_result, gpu_a, gpu_b, ARRAY_SIZE);
	} else {
		printf("Unexpected operation type: %s. Exiting...\n", operation.c_str());
		checkCuda( hipFree(gpu_a) );
		checkCuda( hipFree(gpu_b) );
		checkCuda( hipFree(gpu_result) );
		exit(1);
	}
	checkCuda( hipDeviceSynchronize() );
	auto stop = std::chrono::high_resolution_clock::now();
	std::cout << "Time elapsed GPU = " << std::chrono::duration_cast<chrono::nanoseconds>(stop - start).count() << " ns\n";

	checkCuda( hipGetLastError() );

	// Copy data back and synchronize
	checkCuda( hipMemcpy( cpu_result, gpu_result, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost ) );
	checkCuda( hipDeviceSynchronize() );
	if (verbose) {
		printf("Results of operation: \n");
		for (int i = 0; i < min(5, ARRAY_SIZE); i++) {
			printf("Result[%d]: %d, A[%d]: %d, B[%d], %d\n", i, cpu_result[i], i, cpu_a[i], i, cpu_b[i]);
		}
	}

	checkCuda( hipFree(gpu_a) );
	checkCuda( hipFree(gpu_b) );
	checkCuda( hipFree(gpu_result) );
}


void demonstrateConditionalBranching(int numBlocks, int blockSize, int totalThreads) {
	if (verbose) {
		printf("----- Conditional Branching -----\n");
		printf("Conditional branching kernel, Array length: %d, Array bytes: %d, "
			"Blocks: %d, Threads/block: %d, Total threads: %d\n",
			(int)ARRAY_SIZE, (int)ARRAY_SIZE_IN_BYTES, 
			numBlocks, blockSize, totalThreads);
	}

	initCpuArrays();
	int *gpu_a;
	int *gpu_b;
	int *gpu_result;

	checkCuda( hipMalloc((void **)&gpu_a, ARRAY_SIZE_IN_BYTES) );
	checkCuda( hipMalloc((void **)&gpu_b, ARRAY_SIZE_IN_BYTES) );
	checkCuda( hipMalloc((void **)&gpu_result, ARRAY_SIZE_IN_BYTES) );

	hipMemcpy( gpu_a, cpu_a, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice );
	hipMemcpy( gpu_b, cpu_b, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice );

	// Perform and time the operation, synchronizing before stopping the timer
	auto start = std::chrono::high_resolution_clock::now();
	branchingKernel<<<numBlocks, blockSize>>>(gpu_result, gpu_a, gpu_b, ARRAY_SIZE);
	checkCuda( hipDeviceSynchronize() );
	auto stop = std::chrono::high_resolution_clock::now();
	std::cout << "Time elapsed GPU = " << std::chrono::duration_cast<chrono::nanoseconds>(stop - start).count() << " ns\n";

	checkCuda( hipGetLastError() );

	// Copy data back and synchronize
	checkCuda( hipMemcpy( cpu_result, gpu_result, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost ) );
	checkCuda( hipDeviceSynchronize() );
	checkCuda( hipFree(gpu_a) );
	checkCuda( hipFree(gpu_b) );
	checkCuda( hipFree(gpu_result) );
}


int main(int argc, char** argv)
{

	int totalThreads = (1 << 20);
	int blockSize = 256;  // Also threads / block
	std::string operation("add");

	if (argc >= 2) {
		totalThreads = atoi(argv[1]);
	}
	if (argc >= 3) {
		blockSize = atoi(argv[2]);
	}
	if (argc >= 4) {
		operation = argv[3];
		if (verbose)
			std::cout << "Changed operation to " << operation << "\n";
	}

	int numBlocks = totalThreads/blockSize;

	// validate command line arguments
	if (totalThreads % blockSize != 0) {
		++numBlocks;
		totalThreads = numBlocks*blockSize;
		
		printf("Warning: Total thread count is not evenly divisible by the block size\n");
		printf("The total number of threads will be rounded up to %d\n", totalThreads);
	}

	// printf("Performing warm up run...\n");
	performMathOperations(numBlocks, blockSize, totalThreads, operation);
	
	// printf("Performing real run...\n");
	performMathOperations(numBlocks, blockSize, totalThreads, operation);

	demonstrateConditionalBranching(numBlocks, blockSize, totalThreads);
}
